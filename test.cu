#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 1024

__global__ void add(int *a, int *b, int *c) {
  int tid = threadIdx.x;
  c[tid] = a[tid] + b[tid];
}

int main() {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int i;

  a = (int *)malloc(N * sizeof(int));
  b = (int *)malloc(N * sizeof(int));
  c = (int *)malloc(N * sizeof(int));

  for (i = 0; i < N; i++) {
    a[i] = i;
    b[i] = i;
  }

  hipMalloc((void **)&d_a, N * sizeof(int));
  hipMalloc((void **)&d_b, N * sizeof(int));
  hipMalloc((void **)&d_c, N * sizeof(int));

  hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

  add<<<1, N>>>(d_a, d_b, d_c);

  hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

  for (i = 0; i < N; i++) {
    printf("%d\n", c[i]);
  }

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  free(a);
  free(b);
  free(c);

  return 0;
}